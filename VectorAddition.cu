#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "wb.h"

__global__ void vecAdd(float *in1, float *in2, float *out, int len) {
	int i = threadIdx.x + blockDim.x * blockIdx.x;

	out[i] = in1[i] + in2[i];
}

int main(int argc, char **argv) {
	//vars
	wbArg_t args;
	int inputLength;
	float *hostInput1;
	float *hostInput2;
	float *hostOutput;
	float *deviceInput1;
	float *deviceInput2;
	float *deviceOutput;
	hipError_t cudaStatus;

	//Input args
	args = wbArg_read(argc, argv);
	wbTime_start(Generic, "Importing data and creating memory on host");
	hostInput1 =
		(float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
	hostInput2 =
		(float *)wbImport(wbArg_getInputFile(args, 1), &inputLength);
	hostOutput = (float *)malloc(inputLength * sizeof(float));
	wbTime_stop(Generic, "Importing data and creating memory on host");
	wbLog(TRACE, "The input length is ", inputLength);

	//GPU memory allocation
	wbTime_start(GPU, "Allocating GPU memory.");

	cudaStatus = hipMalloc(&deviceInput1, inputLength * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc(&deviceInput2, inputLength * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc(&deviceOutput, inputLength * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	wbTime_stop(GPU, "Allocating GPU memory.");

	//Memory copy to GPU
	wbTime_start(GPU, "Copying input memory to the GPU.");

	cudaStatus = hipMemcpy(deviceInput1, hostInput1, inputLength * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(deviceInput2, hostInput2, inputLength * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	wbTime_stop(GPU, "Copying input memory to the GPU.");

	// Launch computations
	wbTime_start(Compute, "Performing CUDA computation");

	vecAdd<<<1, 256>>>(deviceInput1, deviceInput2, deviceOutput, inputLength);

	hipDeviceSynchronize();
	wbTime_stop(Compute, "Performing CUDA computation");
	
	//Copying result from device to host
	wbTime_start(Copy, "Copying output memory to the CPU");

	cudaStatus = hipMemcpy(hostOutput, deviceOutput, inputLength * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	wbTime_stop(Copy, "Copying output memory to the CPU");

Error:
	wbTime_start(GPU, "Freeing GPU Memory");

	hipFree(deviceInput1);
	hipFree(deviceInput2);
	hipFree(deviceOutput);

	wbTime_stop(GPU, "Freeing GPU Memory");
	wbSolution(args, hostOutput, inputLength);

	free(hostInput1);
	free(hostInput2);
	free(hostOutput);

	return cudaStatus;
}